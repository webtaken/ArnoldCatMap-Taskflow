#include "hip/hip_runtime.h"
/*
Para compilar el siguiente proyecto colocar el siguiente comando:
nvcc -std=c++17 -I <path to taskflow repo> --extended-lambda --gpu-architecture=compute_<your compute GPU's capability> ArnoldTransform.cu -o ArnoldTransform
*/
// TASKFLOW
#include <taskflow/taskflow.hpp>      // core taskflow routines
#include <taskflow/cuda/cudaflow.hpp> // core cudaflow routines

// STB_IMAGE
#include "stb_image/stb_image.cpp"
#include "stb_image/stb_image_write.cpp"

// STD C++
#include <iostream>
#include <ostream>

class IMG_PROPS
{
public:
    int width, height, no_channels, desired_no_channels;
    IMG_PROPS(int _width, int _height, int _no_channels, int _desired_no_channels){
        width = _width;
        height = _height;
        no_channels = _no_channels;
        desired_no_channels = _desired_no_channels;
    }
    ~IMG_PROPS(){}
};


// Arnold's Cat Map Kernel
__global__ void ArnoldTransformKernel(unsigned char* Pin, unsigned char* Pout, int N, int channels)
{
    int Col = threadIdx.x + blockDim.x * blockIdx.x;
    int Row = threadIdx.y + blockDim.y * blockIdx.y;

    if(Col < N && Row < N){
        // Ver el siguient enlace para ver la fórmula de la transformada de Arnold
        // http://fibonacci.math.uri.edu/~kulenm/diffeqaturi/victor442/index.html
        int newCol = (Col + Row) % N;
        int newRow = (Col + 2*Row) % N;
        
        int offset = (Row * N + Col) * channels;
        int newOffset = (newRow * N + newCol) * channels;

        // Valores RGB 
        Pout[newOffset] = Pin[offset]; // R
        Pout[newOffset + 1] = Pin[offset + 1]; // G
        Pout[newOffset + 2] = Pin[offset + 2]; // B
    }
}

// main function begins
int main(int argc, char **argv)
{
    // se debe proveer el path de la imagen a recibir
    if(argc < 2){
        std::cout << "You must provide and image name\n";
        exit(EXIT_FAILURE);
    }
    // en el primer argumento se lee el nombre del archivo
    std::string img_path(argv[1]);
    std::string img_filename = img_path.substr(0, img_path.find('.'));
    std::string img_extention = img_path.substr(img_path.find('.'));
    
    // if(img_extention != ".jpg"){
    //     std::cout << "Only .jpg images are allowed\n";
    //     exit(EXIT_FAILURE);
    // }

    int width, height, original_no_channels;
    int desired_no_channels = 3; // solo 3 porque solo procesaremos imágenes .jpg
    unsigned char *img = stbi_load(img_path.c_str(), &width, &height,
                                   &original_no_channels, desired_no_channels);
    if (img == NULL)
    {
        printf("Error in loading the image\n");
        exit(EXIT_FAILURE);
    }
    IMG_PROPS img_props(width, height, original_no_channels, desired_no_channels);
    printf("Loaded image characteristics:\n");
    printf("width: %dpx\n", width); 
    printf("height: %dpx\n", height);
    printf("original N channels: %d\n", original_no_channels); 
    printf("loaded with N channels: %d\n", desired_no_channels);

    tf::Taskflow taskflow;
    tf::Executor executor;

    taskflow.name("Arnold's Cat Map Algorithm");

/////////////////////////////////////////////////////////////////////////////////////////////
    // la iteración N-ésima de todo el proceso y el máximo número de iteraciones
    int iter_transform = 0, max_iters = 1000; 

    unsigned char *h_Pin = {nullptr}; // imagen adaptada a dimensión NxN
    unsigned char *h_Pout = {nullptr}; // imagen adaptada resultante de la transformada

    // Imágenes a ser alojadas en la GPU
    unsigned char *d_Pin = {nullptr}; // imagen adaptada en la GPU
    unsigned char *d_Pout = {nullptr}; // imagen resultante de la transformación en la GPU
    bool init1 = true, init2 = true; // flags para una sola reserva en memoria en GPU de d_Pin y d_Pout

    // tamaño de la imagen adaptada NxN con sus respectivos canales
    int N = std::max(width, height);
////////////////////////////////////////////////////////////////////////////////////////////

    // aquí colocaremos las funciones de taskflow
    tf::Task resizer = taskflow.emplace([&]()
    {
        // la imagen adaptada es de NxNxChannels
        h_Pin = new unsigned char[N * N * img_props.desired_no_channels];
        // reservamos memoria para el resultado de la transformación continua
        h_Pout = new unsigned char[N * N * img_props.desired_no_channels];

        if(img_props.width > img_props.height)
        {
            // esta conversión es mas sencilla porque la imagen está en row order
            int i;
            // copiamos la imagen tal cual en la imagen adaptada
            for (i = 0; i < img_props.width * img_props.height * img_props.desired_no_channels; i++)
                h_Pin[i] = img[i];
            // rellenamos el resto de pixeles negros (0s) el resto de la imagen adaptada
            for(; i < N * N * img_props.desired_no_channels; i++)
                h_Pin[i] = (unsigned char)0; // rellenamos con un pixel negro
        }
        else // width <= height
        {
            for (int j = 0; j < N; j++)
            {
                int h = 0;
                for (int i = 0; i < N; i++)
                {
                    int offset = (j * N + i) * img_props.desired_no_channels;
                    int offset_orig = (j * img_props.width + h) * img_props.desired_no_channels;
                    if (offset < ((j * N + img_props.width) * img_props.desired_no_channels))
                    {
                        h_Pin[offset] = img[offset_orig]; // R
                        h_Pin[offset + 1] = img[offset_orig + 1]; // G
                        h_Pin[offset + 2] = img[offset_orig + 2]; // B
                        h++;
                    }
                    else
                    {
                        h_Pin[offset] = (unsigned char)0; // rellenamos con un pixel negro
                    }
                }
            }
        }

        std::string transformed_image_name = img_filename + "_arnold_iter_" 
                                        + std::to_string(iter_transform) + img_extention;
        iter_transform++; // incrementamos la iteración
        // escribimos la imagen resultante en la misma carpeta
        stbi_write_jpg(transformed_image_name.c_str(), img_props.width, img_props.height, 
                                    img_props.no_channels, h_Pin, 100);
    }).name("resizer");

    // una tarea puente
    tf::Task helper = taskflow.emplace(
        [&](){}).name("Helper");

    auto [alloc_Pin, alloc_Pout] = taskflow.emplace(
        [&]()
        { 
            if (init1)
            {
                hipMalloc(&d_Pin, N * N * img_props.desired_no_channels * sizeof(unsigned char)); 
                init1 = false;
            }
        },
        [&]()
        { 
            if(init2) {
                hipMalloc(&d_Pout, N * N * img_props.desired_no_channels * sizeof(unsigned char));
                init2 = false;
            }
        }
    );
    // colocando el nombre de cada proceso de alojación
    alloc_Pin.name("alloc_Pin");
    alloc_Pout.name("alloc_Pout");

    tf::Task arnoldflow = taskflow.emplace([&](tf::cudaFlow &cf) {
        // transfiriendo datos de la imagen adaptada (host) a la memoria en el dispositivo
        tf::cudaTask Pin_h2d = cf.copy(d_Pin, h_Pin, N * N * img_props.desired_no_channels).name("Pin_h2d");
        // transfiriendo el resultado del dispositivo al host
        tf::cudaTask Pout_d2h = cf.copy(h_Pout, d_Pout, N * N * img_props.desired_no_channels).name("Pout_d2h");

        // dimensiones del grid
        dim3 dimGrid(ceil(N / 16.0f), ceil(N / 16.0f), 1);
        // dimensiones de los bloques
        dim3 dimBlock(16, 16, 1);

        // launch ArnoldKernel<<<dimGrid, dimBlock, 0>>>(d_Pin, d_Pout, N, 3)
        tf::cudaTask ArnoldKernel = cf.kernel(
            dimGrid, dimBlock, 0, ArnoldTransformKernel, d_Pin, d_Pout, N, 
            img_props.desired_no_channels).name("ArnoldKernel");

        // construimos el flujo de trabajo
        ArnoldKernel.succeed(Pin_h2d).precede(Pout_d2h);
    }).name("ArnoldTransformCudaFlow");

    // esta es una función condicional
    tf::Task convergence_checker = taskflow.emplace([&]() {
        // se llego al tope de las iteraciones
        if(iter_transform >= max_iters){
            // En este caso se ha llegado a la imagen original, entonces pasamos a la tarea final
            // Antes liberamos la memoria reservada en la GPU de las dos imágenes pasadas
            hipFree(d_Pin);
            hipFree(d_Pout);
            return 1;     
        }

        // compararemos la imagen original img con el resultado de la transformación h_Pout
        for (int j = 0, y = 0; j < img_props.height; j++, y++)
        {
            for (int i = 0; i < img_props.width; i++)
            {
                int offset_orig = (j * img_props.width + i) * img_props.desired_no_channels;
                int offset_transform = (y * N + i) * img_props.desired_no_channels;
                if (h_Pout[offset_transform] != img[offset_orig] ||         // R
                    h_Pout[offset_transform + 1] != img[offset_orig + 1] || // G
                    h_Pout[offset_transform + 2] != img[offset_orig + 2]) // B
                {
                    // copiamos los datos de h_Pout hacia h_Pin para la siguiente iteración
                    memcpy(h_Pin, h_Pout, N * N * img_props.desired_no_channels * sizeof(unsigned char));
                    
                    // antes guardaremos la imagen de la iteración N ésima del proceso
                    // de la transformada 
                    unsigned char *tmp_img = new unsigned char[img_props.width * img_props.height 
                                                        * img_props.desired_no_channels];

                    for(int p = 0; p < img_props.height; p++){
                        for(int q = 0; q < img_props.width; q++){
                            int offset_orig = (p * img_props.width + q) * img_props.desired_no_channels;
                            int offset_transform = (p * N + q) * img_props.desired_no_channels;
                
                            // copiando los valores RGB en la imagen a guardarse
                            tmp_img[offset_orig] = h_Pin[offset_transform]; // R
                            tmp_img[offset_orig + 1] = h_Pin[offset_transform + 1]; // G
                            tmp_img[offset_orig + 2] = h_Pin[offset_transform + 2]; // B
                        }
                    }

                    std::string transformed_image_name = img_filename + "_arnold_iter_" 
                                                        + std::to_string(iter_transform) + img_extention;
                    iter_transform++; // incrementamos la iteración
                    // escribimos la imagen resultante en la misma carpeta
                    stbi_write_jpg(transformed_image_name.c_str(), img_props.width, img_props.height, 
                                    img_props.no_channels, tmp_img, 100);
                    
                    // siempre debemos liberar la memoria
                    delete [] tmp_img;
                    // volvemos a la primera función
                    return 0;
                }
            }
        }

        // copiamos los datos de h_Pout hacia h_Pin para copiar el resultado final
        memcpy(h_Pin, h_Pout, N * N * img_props.desired_no_channels * sizeof(unsigned char));
        // antes guardaremos la última imagen de la iteración N ésima del proceso
        // de la transformada de Arnold
        unsigned char *tmp_img = new unsigned char[img_props.width * img_props.height 
                                                        * img_props.desired_no_channels];

        for(int p = 0; p < img_props.height; p++){
            for(int q = 0; q < img_props.width; q++){
                int offset_orig = (p * img_props.width + q) * img_props.desired_no_channels;
                int offset_transform = (p * N + q) * img_props.desired_no_channels;
                
                // copiando los valores RGB en la imagen a guardarse
                tmp_img[offset_orig] = h_Pin[offset_transform]; // R
                tmp_img[offset_orig + 1] = h_Pin[offset_transform + 1]; // G
                tmp_img[offset_orig + 2] = h_Pin[offset_transform + 2]; // B
            }
        }
                    
        std::string transformed_image_name = img_filename + "_arnold_iter_" 
                                    + std::to_string(iter_transform) + img_extention;
        // escribimos la imagen resultante en la misma carpeta
        stbi_write_jpg(transformed_image_name.c_str(), img_props.width, img_props.height, 
                                img_props.no_channels, tmp_img, 100);

        // volvemos a la primera función
        // En este caso se ha llegado a la imagen original, entonces pasamos a la tarea final
        // Antes liberamos la memoria reservada en la GPU de las dos imágenes pasadas
        hipFree(d_Pin);
        hipFree(d_Pout);
        // siempre debemos liberar la memoria
        delete [] tmp_img;
        
        return 1;     
    }).name("convergence&cls");



    // La última tarea a realizarse
    tf::Task finalizer = taskflow.emplace(
        [&]()
        { 
            std::cout << "Arnold Transformation ended\n";
            std::cout << "Image path: " << img_path << "\n";
            std::cout << "With " << iter_transform << " iterations\n";

            // Terminamos las tareas liberando los recursos empleados
            // siempre debemos liberar la memoria
            stbi_image_free(img);

            // liberamos la memoria de ambas imágenes en CPU
            if (h_Pin != nullptr)
            {
                delete[] h_Pin; // también debemos liberar esta memoria
            }
            if (h_Pout != nullptr)
            {
                delete[] h_Pout; // también debemos liberar esta memoria
            }
        }).name("Finalizer");

//////////////////////////////////////////////////////////////////////////////////////////
    // Ahora construimos el grafo de tareas
    resizer.precede(helper);
    helper.precede(alloc_Pin, alloc_Pout);
    arnoldflow.succeed(alloc_Pin, alloc_Pout);
    arnoldflow.precede(convergence_checker);
    convergence_checker.precede(helper, finalizer);
    
    executor.run(taskflow).wait();

    taskflow.dump(std::cout); // mostramos el grafo como un archivo de graphviz

}